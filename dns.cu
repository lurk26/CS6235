#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <math.h>
#include <time.h>
#include <vector>
#include <iomanip>
#include <algorithm>
#include <string>
#include <map>
#include <helper_image.h>     // helper for image and data comparison
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

#define BLOCK_SIZE 32 // Number of threads in x and y direction - Maximum Number of threads per block = 32 * 32 = 1024

// Kernel Definitions

//__global__ void sobel( int xd_size, int yd_size, int maxdval, int d_thresh, unsigned int *input , int *output)
//{

int main()
{
    float start_clock = clock();
//    ofstream f("lx4 - Re300 - Fr300 - results.txt"); // Solution Results
      ofstream f("result_cpu4.txt"); // Solution Results
    f.setf(ios::fixed | ios::showpoint);
    f << setprecision(5);

    ofstream g("lx4 - Re300 - Fr300 - convergence.txt"); // Convergence history
    g.setf(ios::fixed | ios::showpoint);
    g << setprecision(5);
    cout.setf(ios::fixed | ios::showpoint);
    cout << setprecision(5);

    float Re, Pr, Fr, T_L, T_0, T_amb, dx, dy, t, ny, nx, dt, eps, abs, beta, iter, maxiter, tf, st, pold, counter, column, u_wind, T_R, Lx, Ly;

    // Input parameters 
    Lx = 2 * 2.0; Ly = 5.0; // Domain dimensions
    nx = Lx * 2.0; ny = Ly * 2.0; // Grid size - Number of nodes
    u_wind = 1; // Reference velocity
//    viscosity = 0.5*(16.97 + 18.90)*pow(10.0, -6.0); // Fluid viscosity
    st = 0.00005; // Total variance criteria
    eps = 0.001; // Pressure convergence criteria
    tf = 100; // Final time step
    Pr = 0.5*(0.709 + 0.711); // Prandtl number
    Re = 30.0; Fr = 0.3; // Non-dimensional numbers for inflow conditions
    dx = Lx / (nx - 1); dy = Ly / (ny - 1); // dx and dy
    beta = 1.4; // Successive over relaxation factor (SOR)
    t = 0; // Initial time step
    T_L = 100.0; // Left wall temperature (C)
    T_R = 50.0; // Right wall temperature (C)
    T_amb = 25.0; // Ambient air temperature (C)
    T_0 = 50.0; // Initial air temperature
    T_L = T_L + 273.15; T_0 = T_0 + 273.15; T_amb = T_amb + 273.15; T_R = T_R + 273.15;// Unit conversion to (K)
    maxiter = 100; // Maximum iteration at each time step
    counter = 0; // initial row for output monitoring
    column = 1; // Column number for output display

    // Records number of clicks a step takes
    std::map<string, uint32_t> stepTimingAccumulator;



//.................................................GPU Version.................................................


thrust::host_vector<float> u(nx*(ny+1));
thrust::host_vector<float> us(nx*(ny+1));
thrust::host_vector<float> uold(nx*(ny+1));

thrust::host_vector<float> v((nx+1)*ny);
thrust::host_vector<float> vs((nx+1)*ny);
thrust::host_vector<float> vold((nx+1)*ny);

thrust::host_vector<float> p((nx+1)*(ny+1));
thrust::host_vector<float> T((nx+1)*(ny+1));
thrust::host_vector<float> Told((nx+1)*(ny+1));

thrust::host_vector<float> sai(nx*ny);
thrust::host_vector<float> omc_gpu(nx*ny);
thrust::host_vector<float> vc_gpu(nx*ny);
thrust::host_vector<float> uc_gpu(nx*ny);

thrust::host_vector<float> pc_gpu(nx*ny);
thrust::host_vector<float> Tc_gpu(nx*ny);

thrust::device_vector<float> u_h(nx*(ny+1));
thrust::device_vector<float> us_h(nx*(ny+1));
thrust::device_vector<float> uold_h(nx*(ny+1));

thrust::device_vector<float> v_h((nx+1)*ny);
thrust::device_vector<float> vs_h((nx+1)*ny);
thrust::device_vector<float> vold_h((nx+1)*ny);

thrust::device_vector<float> p_h((nx+1)*(ny+1));
thrust::device_vector<float> T_h((nx+1)*(ny+1));
thrust::device_vector<float> Told_h((nx+1)*(ny+1));

thrust::device_vector<float> sai_h(nx*ny);
thrust::device_vector<float> omc_h_gpu(nx*ny);
thrust::device_vector<float> vc_h_gpu(nx*ny);
thrust::device_vector<float> uc_h_gpu(nx*ny);

thrust::device_vector<float> pc_h_gpu(nx*ny);
thrust::device_vector<float> Tc_h_gpu(nx*ny);


int wu, wv, wp, wT, wc;

wu = nx; // number of rows of u vector
wv = nx + 1; // number of rows of v vector
wp = nx + 1; // number of rows of p vector
wT = nx + 1; // number of rows of T vector
//wsai = nx; // number of rows of sai vector
wc = nx; // number of rows of collocated vectors

/*  
    vector<vector<float> > u(nx, vector<float>(ny + 1));
    vector<vector<float> > us(nx, vector<float>(ny + 1));
    vector<vector<float> > uold(nx, vector<float>(ny + 1));

    vector<vector<float> > v(nx + 1, vector<float>(ny));
    vector<vector<float> > vs(nx + 1, vector<float>(ny));
    vector<vector<float> > vold(nx + 1, vector<float>(ny));

    vector<vector<float> > p(nx + 1, vector<float>(ny + 1));
    vector<vector<float> > T(nx + 1, vector<float>(ny + 1));
    vector<vector<float> > Told(nx + 1, vector<float>(ny + 1));

    vector<vector<float> > sai(nx, vector<float>(ny));
    vector<vector<float> > omc_cpu(nx, vector<float>(ny));
    vector<vector<float> > vc_cpu(nx, vector<float>(ny));
    vector<vector<float> > uc_cpu(nx, vector<float>(ny));

    vector<vector<float> > pc_cpu(nx, vector<float>(ny));
    vector<vector<float> > Tc_cpu(nx, vector<float>(ny));
*/


    // Time step size stability criterion

    float mt1 = 0.25*pow(dx, 2.0) / (1.0 / Re); float Rer = 1.0 / Re; float mt2 = 0.25*pow(dy, 2.0) / (1.0 / Re);

    if (mt1 > Rer)
    {
        dt = Rer;
    }
    else
    {
        dt = mt1;
    }

    if (dt > mt2)
    {
        dt = mt2;
    }

    
    //......................................................................................
    // Step 0 - It can be parallelized
    // Initializing the flow variable (Temperature)  
    // Boundary conditions for T (Initialization)
    int step0_start = clock();
    for (int i = 0; i < nx + 1; i++)
    {
        for (int j = 0; j < ny + 1; j++)
        {
            T[i * wT + j] = T_0 / T_amb;
        } // end for j
    } // end for i
    //......................................................................................
    int step0_end = clock();
    stepTimingAccumulator["Step 0, Initializing Temperature"] += step0_end - step0_start;
    //......................................................................................

    // Marching in Time - Outermost loop

    while (t <= tf)
    {

        iter = 0;

        int stepi1_start = clock();
        //........................................................................................
        // Step i1 - it can be parallelized 
        // boundary conditions for u velocity

        for (int i = 0; i < nx; i++)
        {
            for (int j = 0; j < ny + 1; j++)
            {
                if (i == 0 && j > 0 && j < ny)
                {
                    if (j*dy < 2.0)
                    {
                        u[i * wu + j] = 0; // left wall - Final
                    }
                    else
                    {
                        u[i * wu + j] = u_wind; // left inlet - Final
                    }
                }
                else if (i == nx - 1 && j>0 && j < ny)
                {
                    if (j*dy < 2.0)
                    {
                        u[i * wu + j] = 0; // Right wall has 0 horizontal velocity - Final
                    }
                    else
                    {
                        u[i * wu + j] = u[(i - 1) * wu + j]; // right outlet - no velocity change
                    }
                }
                else if (j == 0)
                {
                    u[i * wu + j] = -u[i * wu + j + 1]; // bottom ghost - Final
                }
                else if (j == ny)
                {
                    u[i * wu + j] = u[i * wu + j - 1]; // upper ghost - Final
                }
            } // end for j
        } // end for i
        int stepi1_end = clock();
        stepTimingAccumulator["Step i1 - Set Horizontal Velocity Boundary Conditions"] += stepi1_end - stepi1_start;
        //...............................................................................................

        
        //.........................................................................................
        // Step i2 - it can be parallelized
        // boundary conditions for v velocity
        int stepi2_start = clock();

        for (int i = 0; i < nx + 1; i++)
        {
            for (int j = 0; j < ny; j++)
            {
                if (j == 0 && i > 0 && i < nx)
                {
                    v[i * wv + j] = 0; // bottom wall - Final
                }
                else if (j == ny - 1 && i > 0 && i < nx)
                {
                    v[i * wv + j] = v[i * wv + j - 1]; // upper wall - Final
                }
                else if (i == 0)
                {
                    v[i * wv + j] = -v[(i + 1) * wv + j]; // left ghost (Left Wall and inlet has 0 vertical velocity) - Final
                }
                else if (i == nx)
                {
                    if (j*dy < 2.0)
                    {
                        v[i * wv + j] = -v[(i -1) * wv + j]; // right ghost (Right wall has 0 vertical velocity) - Final
                    }
                    else
                    {
                        v[i * wv + j] = v[(i - 1) * wv + j]; // right outlet - no velocity gradient
                    }
                }
            } // end for j
        } // end for I
        int stepi2_end = clock();
        stepTimingAccumulator["Step i2 - Set Vertical Velocity Boundary Conditions"] += stepi2_end - stepi2_start;
        //...............................................................................................

        //...............................................................................................
        int step1_start = clock();
        //.........................................................................................
        // Step 1 - it can be parallelized - Solve for intermediate velocity values

        // u - us - vh - a 

        for (int i = 1; i < nx - 1; i++)
        {
            for (int j = 1; j < ny; j++)
            {
                float vh = 1.0 / 4.0*(v[i * wv + j] + v[(i + 1) * wv + j] + v[i * wv + j - 1] + v[(i + 1) * wv + j - 1]); // v hat
                float a = u[i * wu + j] * 1.0 / (2.0*dx)*(u[(i + 1) * wu + j] - u[(i - 1) * wu + j]) + vh*1.0 / (2.0*dy)*(u[i * wu + j + 1] - u[i * wu + j - 1]); // a
                us[i * wu + j] = dt / Re*(1.0 / pow(dx, 2.0)*(u[(i + 1) * wu + j] - 2.0*u[i * wu + j] + u[(i - 1) * wu + j]) + 1.0 / pow(dy, 2.0)*(u[i * wu + j + 1] - 2.0*u[i * wu + j] + u[i * wu + j - 1])) - a*dt + u[i * wu + j]; // u star
            } // end for j
        } // end for i

        //..........................................................................................
        // Step 1 - it can be parallelized
        // v - vs - uh - b
        for (int i = 1; i < nx; i++)
        {
            for (int j = 1; j < ny - 1; j++)
            {
                float uh = 1.0 / 4.0*(u[i * wu + j] + u[i * wu + j + 1] + u[(i - 1) * wu + j] + u[(i - 1) * wu + j + 1]);
                float b = uh*1.0 / (2.0*dx)*(v[(i + 1) * wv + j] - v[(i - 1) * wv + j]) + v[i * wv + j] * 1.0 / (2.0*dy)*(v[i * wv + j + 1] - v[i * wv + j - 1]); // b
                vs[i * wv + j] = dt / Re*(1.0 / pow(dx, 2.0)*(v[(i + 1) * wv + j] - 2.0*v[i * wv + j] + v[(i - 1) * wv + j]) + 1.0 / pow(dy, 2.0)*(v[i * wv + j + 1] - 2.0*v[i * wv + j] + v[i * wv + j - 1])) + dt / pow(Fr, 2.0)*(0.5*(T[i * wT + j] + T[i * wT + j - 1]) - 1) / (0.5*(T[i * wT + j] + T[i * wT + j - 1])) - b*dt + v[i * wv + j]; // v 
            } // end for j
        } // end for i

        //...........................................................................................
        // vs and us on Boundary conditions

        for (int i = 0; i < nx; i++)
        {
            us[i * wu + 0] = -us[i * wu + 1]; // bottom ghost - Final
        } // end for j

        //...........................................................................................
        for (int j = 0; j < ny + 1; j++)
        {
            if (j*dy < 2.0)
            {
                us[0 * wu + j] = 0; // left wall - FInal
                us[(nx - 1) * wu + j] = 0; // right wall - Final
            }
            else
            {
                us[0 * wu + j] = u_wind; // left inlet - Final
            }
        }
        //...........................................................................................

        for (int j = 0; j < ny; j++)
        {
            vs[0 * wv + j] = -vs[1 * wv + j]; // left ghost (Both wall and inlet have 0 vs) - Final
            if (j*dy < 2.0)
            {
                vs[nx * wv + j] = -vs[(nx - 1) * wv + j]; // right ghost (Only the right wall - Final
            }
            else
            {
                vs[nx * wv + j] = vs[(nx - 1) * wv + j]; // right outlet - no flux
            }
        }
        //............................................................................................

        for (int i = 0; i < nx + 1; i++)
        {
            vs[i * wv + 0] = 0; // Bottom wall - Final
        } // end for i
        //............................................................................................

        int step1_end = clock();
        stepTimingAccumulator["Step 1 - Solve for intermediate velocities"] += step1_end - step1_start;

        //...............................................................................................
        // Step 2 - It can be parallelized 
        // This is the most expensive part of the code
        // Poisson equation for pressure
        int step2_start = clock();

        float error = 1; iter = 0;

	thrust::device_vector<float> (nx*ny);


        // Solve for pressure iteratively until it converges - Using Gauss Seidel SOR 
        while (error > eps)
        {
            error = 0;
            //............................................................................................
            for (int i = 1; i < nx; i++)
            {
                for (int j = 1; j < ny; j++)
                {
                    pold = p[i * wp + j];
                    p[i * wp + j] = beta*pow(dx, 2.0)*pow(dy, 2.0) / (-2.0*(pow(dx, 2.0) + pow(dy, 2.0)))*(-1.0 / pow(dx, 2.0)*(p[(i + 1) * wp + j] + p[(i - 1) * wp + j] + p[i * wp + j + 1] + p[i * wp + j - 1]) + 1.0 / dt*(1.0 / dx*(us[i * wu + j] - us[(i - 1) * wu + j]) + 1.0 / dy*(vs[i * wv + j] - vs[i * wv + j - 1]))) + (1.0 - beta)*p[i * wp + j];
                    abs = pow((p[i * wp + j] - pold), 2.0);
                    error = error + abs;
                } // end for j
            } // end for i
            //............................................................................................
            // boundary conditions for pressure

            for (int i = 0; i < nx + 1; i++)
            {
                for (int j = 0; j < ny + 1; j++)
                {
                    if (j == 0)
                    {
                        p[i * wp + j] = p[i * wp + j + 1]; // bottom wall - Final
                    }
                    else if (j == ny)
                    {
                        p[i * wp + j] = p[i * wp + j - 1]; // Upper - no flux
                    }
                    else if (i == 0)
                    {
                        if (j*dy < 2.0)
                        {
                            p[i * wp + j] = p[(i + 1) * wp + j]; // left wall - not the inlet - Final
                        }
                        else
                        {
                            p[i * wp + j] = p[(i + 1) * wp + j];
                        }
                    }
                    else if (i == nx)
                    {
                        if (j*dy < 2.0)
                        {
                            p[i * wp + j] = p[(i - 1) * wp + j]; // right wall - not the outlet - Final
                        }
                        else
                        {
                            p[i * wp + j] = -p[(i - 1) * wp + j]; // pressure outlet - static pressure is zero - Final
                        }
                    }
                } // end for j
            } // end for i
            //................................................................................................

            error = pow(error, 0.5);
            iter = iter + 1;
            if (iter > maxiter)
            {
                break;
            }

        } // end while eps

        int step2_end = clock();
        stepTimingAccumulator["Step 2 - Solve for pressure until tolerance or max iterations"] += step2_end - step2_start;
        //...............................................................................................

        //.................................................................................................
        // Step 3 - It can be parallelized 
        // velocity update - projection method
        int step3_start = clock();

        // u

        for (int i = 1; i < nx - 1; i++)
        {
            for (int j = 1; j < ny; j++)
            {
                uold[i * wu + j] = u[i * wu + j];
                u[i * wu + j] = us[i * wu + j] - dt / dx*(p[(i + 1) * wp + j] - p[i * wp + j]);
            } // end for j
        } // end for i
        //................................................

        // v

        for (int i = 1; i < nx; i++)
        {
            for (int j = 1; j < ny - 1; j++)
            {
                vold[i * wv + j] = v[i * wv + j];
                v[i * wv + j] = vs[i * wv + j] - dt / dy*(p[i * wp + j + 1] - p[i * wp + j]);
            } // end for j
        } // end for i
        int step3_end = clock();
        stepTimingAccumulator["Step 3 - Velocity Update"] += step3_end - step3_start;
        //...............................................................................................

        //...............................................................................................
        // Step 4 - It can be parallelized
        // Solving for temperature
        int step4_start = clock();
        for (int i = 1; i < nx; i++)
        {
            for (int j = 1; j < ny; j++)
            {
                Told[i * wT + j] = T[i * wT + j];
                T[i * wT + j] = T[i * wT + j] + dt*(-0.5*(u[i * wu + j] + u[(i - 1) * wu + j])*(1.0 / (2.0*dx)*(T[(i + 1) * wT + j] - T[(i - 1) * wT + j])) - 0.5*(v[i * wv + j] + v[i * wv + j - 1])*(1.0 / (2.0*dy)*(T[i * wT + j + 1] - T[i * wT + j - 1])) + 1 / (Re*Pr)*(1 / pow(dx, 2.0)*(T[(i + 1) * wT + j] - 2.0*T[i * wT + j] + T[(i - 1) * wT + j]) + 1 / pow(dy, 2.0)*(T[i * wT + j + 1] - 2 * T[i * wT + j] + T[i * wT + j - 1])));
            } // end for j
        } // end for i
        int step4_end = clock();
        stepTimingAccumulator["Step 4 - Solving for temperature"] += step4_end - step4_start;
        //................................................................................................
        
        //...............................................................................................
        // Step i3 - Initializing boundary conditions for temperature 
        // boundary conditions for Temperature
        int stepi3_start = clock();

        for (int i = 0; i < nx + 1; i++)
        {
            for (int j = 0; j < ny + 1; j++)
            {
                if (j == 0)
                {
                    T[i * wT + j] = T[i * wT + j + 1]; // bottom wall - Insulated - no flux - Final
                }
                else if (j == ny)
                {
                    T[i * wT + j] = 2.0*(T_0) / T_amb - T[i * wT + j - 1]; // upper boundary - lid with ambient temperature (as air) - Final
                }
                else if (i == 0)
                {
                    if (j*dy < 2.0)
                    {
                        T[i * wT + j] = 2.0*T_L / T_amb - T[(i + 1) * wT + j]; // left wall at T_L - Constant Temperature - Final
                    }
                    else
                    {
                        T[i * wT + j] = 2.0*T_0 / T_amb - T[(i + 1) + j]; // left inlet at T_0 (initial temperature) - Final
                    }
                }
                else if (i == nx)
                {
                    if (j*dy < 2.0)
                    {
                        T[i * wT + j] = 2.0*T_R / T_amb - T[(i - 1) * wT + j]; // right wall at T_R - Final
                    }
                }
            } // end for j
        } // end for i
        int stepi3_end = clock();
        stepTimingAccumulator["Step i3 - Initializing boundary conditions for temperature"] += stepi3_end - stepi3_start;
        //...............................................................................................

        //...............................................................................................
        // Step 5 - Checking if solution reached steady state
        // Checking the steady state condition
        int step5_start = clock();

        float TV, abs; TV=0; // float abs, TVt, TV2, TV3; TV = 0; TV2 = 0; TV3 = 0; float abs, abs2, abs3;
        for (int i = 1; i < nx - 1; i++)
        {
            for (int j = 1; j < ny - 2; j++)
            {
                abs = v[i * wv + j] - vold[i * wv + j];
                TV = TV + pow(pow(abs, 2), 0.5);
            } // end for i
        } // end for j

        TV = TV / ((nx - 1)*(ny - 2));
	
	float st_time;
        if (TV < st && error < eps)
        {
            st_time = t;
            break;
        }
        counter = counter + 1;
        if (fmod(counter, 10) == 0 || counter == 1)
        {
            //cout << "" << endl;
            //cout << "Column" << setw(30) << "time(s)" << setw(30) << "Iterations on Pressure" << setw(30) << "Pressure Residual" << setw(30) << "Total Variance" << endl;
        } // end if
        int step5_end = clock();
        stepTimingAccumulator["Step 5 - Check for steady state"] += step5_end - step5_start;
        //...............................................................................................


        //cout << column << setw(30) << t << setw(30) << iter << setw(30) << error << setw(30) << TV << endl;
        g << column << setw(30) << t << setw(30) << iter << setw(30) << error << setw(30) << TV << endl;
        t = t + dt;
        column = column + 1;

    } // end while time

    //........................................................................................................

    // Step 6
    // Co-locate the staggered grid points 
    int step6_start = clock();
    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            vc_gpu[i * wc + j] = 1.0 / 2.0*(v[(i + 1) * wv + j] + v[i * wv + j]);
            pc_gpu[i * wc + j] = 1.0 / 4.0*(p[i * wp + j] + p[(i + 1) * wp + j] + p[i * wp + j + 1] + p[(i + 1) * wp + j + 1]);
            uc_gpu[i * wc + j] = 1.0 / 2.0*(u[i * wu + j] + u[i * wu + j + 1]);
            omc_gpu[i * wc + j] = 1.0 / dx*(v[(i + 1) * wv + j] - v[i * wv + j]) - 1.0 / dy*(u[i * wu + j + 1] - u[i * wu + j]);
            Tc_gpu[i * wc + j] = 1.0 / 4.0*(T[i * wT + j] + T[(i + 1) * wT + j] + T[i * wT + j + 1] + T[(i + 1) * wT + j + 1]);

        } // end for j
    } // end for i
    //........................................................................................................
    int step6_end = clock();
    stepTimingAccumulator["Step 6 - Co-locate staggered grid points"] += step6_end - step6_start;

    // Steady state results

    for (int j = 0; j < ny; j++)
    {
        for (int i = 0; i < nx; i++)
        {
            f << setw(15) << t - dt << setw(15) << i*dx << setw(15) << j*dy << setw(15) << uc_gpu[i * wc + j] << setw(15) << vc_gpu[i * wc + j] << setw(15) << pc_gpu[i * wc + j] << setw(15) << Tc_gpu[i * wc + j] * T_amb - 273.15 << setw(15) << omc_gpu[i * wc + j] << endl;
        } // end for i
    } // end for j
    //.........................................................................................................

    float end_clock = clock();

    for (auto it = stepTimingAccumulator.begin(); it != stepTimingAccumulator.end(); it++)
    {
        float seconds = (float)it->second / CLOCKS_PER_SEC;
        std::cout << it->first << "\t" << seconds << endl;
    }
//.................................................End of GPU Version..........................................


// Compare CPU and GPU results
/*  bool success = true; 
 for (int j = 0; j < ny; j++)
  {
      for (int i = 0; i < nx; i++)
      {
	  if (uc_cpu[i][j] != uc_cpu[i][j] || vc_cpu[i][j] != vc_cpu[i][j] || pc_cpu[i][j] != pc_cpu[i][j] || Tc_cpu[i][j] != Tc_cpu[i][j])
          {
             success = false;
          }
      }
 }
	if (success == true)
	{ 
	   printf("\n");
	   printf("*** kernel PASSED ***\n"); //, kernelName);
	   printf("The outputs of CPU version and GPU version are identical.\n");
	}
	else
	{  
	   printf("\n");
	   printf("*** kernel FAILED ***\n"); //, kernelName);
	}
*/
    cout << "" << endl;
    cout << "Steady state time = " << t << " (s) " << endl;
    cout << "GPU time = " << (end_clock - start_clock) / CLOCKS_PER_SEC << " (s)" << endl;

    return 0;
} // end main
